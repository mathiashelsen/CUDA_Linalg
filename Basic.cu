#include "hip/hip_runtime.h"
#include "Basic.h"
#include "Basic.cuh"

__global__ void MatAdd(Matrix A, Matrix B, Matrix C)
{
    if( A.rows != B.rows)
	return;
    if( A.cols != B.cols)
	return;

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y; 
    if( (col < C.cols) && (row < C.rows) )
	C.elems[row + col*C.rows] = A.elems[row + col*A.rows] + B.elems[row + col*B.rows];
}

void GPUMatAdd(Matrix A, Matrix B, Matrix C)
{
    int sizeA = A.rows * A.cols * sizeof(float);

    Matrix a(A.rows, A.cols, MemoryLocationGPU), b(B.rows, B.cols, MemoryLocationGPU), c(C.rows, C.cols, MemoryLocationGPU);
    hipMalloc( &(a.elems), sizeA );
    hipMalloc( &(b.elems), sizeA );
    hipMalloc( &(c.elems), sizeA );

    hipMemcpy( a.elems, A.elems, sizeA, hipMemcpyHostToDevice );
    hipMemcpy( b.elems, B.elems, sizeA, hipMemcpyHostToDevice );

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(A.cols/threadsPerBlock.x + 1, A.rows/threadsPerBlock.y + 1);

    MatAdd<<<numBlocks, threadsPerBlock >>>(a, b, c);
    
    hipMemcpy( C.elems, c.elems, sizeA, hipMemcpyDeviceToHost ); 
    
    hipFree( a.elems );
    hipFree( b.elems );
    hipFree( c.elems );

}

void CPUMatAdd(Matrix A, Matrix B, Matrix C)
{
    int i = 0, j = 0; 
    for( i = 0; i < A.cols; i++ )
    {
	for( j = 0; j < A.rows; j++ )
	{
	    C.elems[j + i*A.rows] = A.elems[j + i*A.rows] + B.elems[j + i*A.rows];
	}
    }
}
